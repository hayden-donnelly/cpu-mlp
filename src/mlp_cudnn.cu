#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <cstdio>
#include "mnist.hpp"

#define CHECK_CUDNN(expression) \
{ \
    hipdnnStatus_t status = (expression); \
    if (status != HIPDNN_STATUS_SUCCESS) \
    { \
        std::cerr << "Error on line " << __LINE__ << ": " \
            << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
}

int main()
{
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));
    printf("Initialized cuDNN\n");

    load_mnist();
    print_image(train_image[2]);

    constexpr int input_dim = 784;
    constexpr int hidden_dim = 256;
    constexpr int output_dim = 10;
}
