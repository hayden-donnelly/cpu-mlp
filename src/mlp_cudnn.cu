#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <cstdio>
#include "mnist.hpp"

#define CHECK_CUDNN(expression) \
{ \
    hipdnnStatus_t status = (expression); \
    if (status != HIPDNN_STATUS_SUCCESS) \
    { \
        std::cerr << "Error on line " << __LINE__ << ": " \
            << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
}

int main()
{
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));
    printf("Initialized cuDNN\n");

    load_mnist();
    printf("Loaded MNIST\n");
    print_image(train_image[2]);

    constexpr int input_dim = 784;
    constexpr int hidden_dim = 256;
    constexpr int output_dim = 10;
    constexpr int batch_size = 32;

    // Setup input, output, and activation descriptors.
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t hidden_desc;
    hipdnnTensorDescriptor_t output_desc;

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&hidden_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));

    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, input_dim
    ));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        hidden_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, hidden_size
    ));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, output_size
    ));
    
    CHECK_CUDNN(hipdnnDestroy(cudnn));
}
