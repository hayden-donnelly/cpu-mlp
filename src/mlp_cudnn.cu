#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <cstdio>
#include "mnist.hpp"

#define CHECK_CUDNN(expression) \
{ \
    hipdnnStatus_t status = (expression); \
    if(status != HIPDNN_STATUS_SUCCESS) \
    { \
        std::cerr << "Error on line " << __LINE__ << ": " \
            << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUDA(expression) \
{ \
    hipError_t error = (expression); \
    if(error != 0) \
    { \
        std::cerr << "Error on line " << __LINE__ << ": " \
            << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
}

int main()
{
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));
    printf("Initialized cuDNN\n");

    load_mnist();
    printf("Loaded MNIST\n");
    print_image(train_image[2]);

    constexpr int input_dim = 784;
    constexpr int hidden_dim = 256;
    constexpr int output_dim = 10;
    constexpr int batch_size = 32;

    // Setup input, output, and hidden descriptors.
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t hidden_desc;
    hipdnnTensorDescriptor_t output_desc;

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&hidden_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));

    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, input_dim
    ));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        hidden_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, hidden_dim
    ));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, 1, output_dim
    ));
 
    // Create fully connected layer descriptors.
    hipdnnTensorDescriptor_t fc1_filter_desc;
    hipdnnTensorDescriptor_t fc2_filter_desc;

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&fc1_filter_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&fc2_filter_desc));

    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        fc1_filter_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, hidden_dim, 1, input_dim, 1 
    ));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        fc2_filter_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_dim, 1, hidden_dim, 1 
    ));

    // Setup ReLU.
    hipdnnActivationDescriptor_t relu_desc;
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&relu_desc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(
        relu_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0
    ));

    // Setup softmax descriptor.
    // HIPDNN_SOFTMAX_ACCURATE represents "safe softmax",
    // (max value is subtracted from exponents to prevent overflow).
    hipdnnSoftmaxAlgorithm_t softmax_algo = HIPDNN_SOFTMAX_ACCURATE;
    hipdnnSoftmaxMode_t softmax_mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
    
    // Allocate memory for weights and biases.
    float* d_fc1_weights;
    float* d_fc2_weights;
    float* d_fc1_bias;
    float* d_fc2_bias;
    float* d_input;
    float* d_hidden;
    float* d_output;

    CHECK_CUDA(hipMalloc(&d_fc1_weights, sizeof(float) * input_dim * hidden_dim));
    CHECK_CUDA(hipMalloc(&d_fc2_weights, sizeof(float) * hidden_dim * output_dim));
    CHECK_CUDA(hipMalloc(&d_fc1_bias, sizeof(float) * hidden_dim));
    CHECK_CUDA(hipMalloc(&d_fc2_bias, sizeof(float) * output_dim));
    CHECK_CUDA(hipMalloc(&d_input, sizeof(float) * batch_size * input_dim));
    CHECK_CUDA(hipMalloc(&d_hidden, sizeof(float) * batch_size * hidden_dim));
    CHECK_CUDA(hipMalloc(&d_output, sizeof(float) * batch_size * output_dim));

    // Define forward pass. 
    float alpha = 1.0f, beta = 0.0f;

    // First fully connected layer + ReLU.
    CHECK_CUDNN(cudnnFullyConnectedForward(
        cudnn,
        &alpha,
        input_desc, d_input,
        fc1_filter_desc, d_fc1_weights,
        d_fc1_bias,
        &beta,
        hidden_desc, d_hidden
    ));

    CHECK_CUDNN(hipdnnDestroy(cudnn));
}
